#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <memory>
#include <algorithm>
#include "BMPImage.hpp"
#include "GaussianBlur.hpp"

#define AssertCuda(error_code)                                           \
    if (error_code != hipSuccess)                                       \
    {                                                                    \
        std::cout << "The cuda call in " << __FILE__ << " on line "      \
                  << __LINE__ << " resulted in the error '"              \
                  << hipGetErrorString(error_code) << "'" << std::endl; \
        std::abort();                                                    \
    }

// Templated Laplacian function
template <typename Number>
void applyLaplacian(const unsigned char *input, std::vector<Number> &output, int width, int height)
{
    const int kernel[3][3] = {
        {0, 1, 0},
        {1, -4, 1},
        {0, 1, 0}};

    for (int y = 1; y < height - 1; ++y)
    {
        for (int x = 1; x < width - 1; ++x)
        {
            Number sum = 0.0f;
            for (int ky = -1; ky <= 1; ++ky)
            {
                for (int kx = -1; kx <= 1; ++kx)
                {
                    sum += input[(y + ky) * width + (x + kx)] * kernel[ky + 1][kx + 1];
                }
            }
            output[y * width + x] = sum;
        }
    }
}

// Templated CUDA kernel for gradient descent
template <typename Number>
__global__ void gradDescCuda(Number *x, const Number *b, const Number *laplacian, Number mu, Number learningRate, int width, int height, int maxIterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalSize = width * height;

    if (idx < totalSize)
    {
        for (int iter = 0; iter < maxIterations; ++iter)
        {
            Number gradient = (x[idx] - b[idx]) + mu * laplacian[idx] * x[idx];
            x[idx] -= learningRate * gradient;
        }
    }
}

// Templated function to solve minimization problem using CUDA
template <typename Number>
void solveMinCuda(std::vector<Number> &x, const std::vector<Number> &b, const std::vector<Number> &laplacian, Number mu, int width, int height)
{
    int totalSize = width * height;
    Number learningRate = 0.001f;
    int maxIterations = 100;
    int blockSize = 256;
    int numBlocks = (totalSize + blockSize - 1) / blockSize;

    // Allocate device memory
    Number *d_x, *d_b, *d_laplacian;
    AssertCuda(hipMalloc(&d_x, totalSize * sizeof(Number)));
    AssertCuda(hipMalloc(&d_b, totalSize * sizeof(Number)));
    AssertCuda(hipMalloc(&d_laplacian, totalSize * sizeof(Number)));

    // Copy data from host to device
    AssertCuda(hipMemcpy(d_x, x.data(), totalSize * sizeof(Number), hipMemcpyHostToDevice));
    AssertCuda(hipMemcpy(d_b, b.data(), totalSize * sizeof(Number), hipMemcpyHostToDevice));
    AssertCuda(hipMemcpy(d_laplacian, laplacian.data(), totalSize * sizeof(Number), hipMemcpyHostToDevice));

    // Launch the CUDA kernel
    gradDescCuda<<<numBlocks, blockSize>>>(d_x, d_b, d_laplacian, mu, learningRate, width, height, maxIterations);
    AssertCuda(hipGetLastError());

    // Copy result back to host
    AssertCuda(hipMemcpy(x.data(), d_x, totalSize * sizeof(Number), hipMemcpyDeviceToHost));

    // Free device memory
    AssertCuda(hipFree(d_x));
    AssertCuda(hipFree(d_b));
    AssertCuda(hipFree(d_laplacian));
}

int main()
{
    using Number = float; // You can change this to double if needed

    auto image = std::make_shared<BMPImage>("lena.bmp");
    int width = image->GetWidth();
    int height = image->GetHeight();
    int kernelSize = 7;
    Number sigma = 5.0f;
    auto blurredImage = std::unique_ptr<unsigned char[]>(new unsigned char[width * height]);
    applyGaussianBlur(image->GetData(), blurredImage.get(), width, height, kernelSize, sigma);

    BMPImage blurred(width, height, blurredImage.get());
    blurred.SaveBMP("lena_blurred.bmp");

    std::vector<Number> laplacian(width * height, 0.0f);
    applyLaplacian(blurredImage.get(), laplacian, width, height);

    std::vector<Number> b(width * height, 0.0f);
    for (int i = 0; i < width * height; ++i)
    {
        b[i] = static_cast<Number>(blurredImage[i]);
    }

    std::vector<Number> output(width * height, 0.0f);
    Number mu = 1.0f;

    // Call the templated CUDA function
    solveMinCuda(output, b, laplacian, mu, width, height);

    // Convert output to an image format
    auto outputImage = std::unique_ptr<unsigned char[]>(new unsigned char[width * height]);
    Number minVal = *std::min_element(output.begin(), output.end());
    Number maxVal = *std::max_element(output.begin(), output.end());

    for (int i = 0; i < width * height; ++i)
    {
        outputImage[i] = static_cast<unsigned char>(std::min(std::max(int((output[i] - minVal) / (maxVal - minVal) * 255), 0), 255));
    }

    BMPImage result(width, height, outputImage.get());
    result.SaveBMP("lena_output_cuda.bmp");

    return 0;
}
